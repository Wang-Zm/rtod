#include "hip/hip_runtime.h"
#include <optix.h>
#include <sutil/vec_math.h>
#include "optixScan.h"

__global__ void kGenAABB_t (
      double3* points,
      double radius,
      unsigned int N,
      OptixAabb* aabb
)
{
  unsigned int particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
  if (particleIndex >= N) return;

  double3 center = points[particleIndex];

  // float3 m_min = center - radius;
  // float3 m_max = center + radius;
  float3 m_min;
  float3 m_max;
  m_min.x = center.x - radius;
  m_min.y = center.y - radius;
  m_min.z = center.z - radius;
  m_max.x = center.x + radius;
  m_max.y = center.y + radius;
  m_max.z = center.z + radius;

  aabb[particleIndex] =
  {
    m_min.x, m_min.y, m_min.z,
    m_max.x, m_max.y, m_max.z
  };
}

extern "C" void kGenAABB(double3* points, double width, unsigned int numPrims, OptixAabb* d_aabb) {
  unsigned int threadsPerBlock = 64;
  unsigned int numOfBlocks = numPrims / threadsPerBlock + 1;

  kGenAABB_t <<<numOfBlocks, threadsPerBlock>>> (
    points,
    width,
    numPrims,
    d_aabb
    );
}
